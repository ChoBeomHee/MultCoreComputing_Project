#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20
#define DIM 2048
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    __device__ float hit(float ox, float oy, float* n) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void kernel(Sphere* s, unsigned char* ptr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * DIM;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    // 이미지 픽셀 값을 설정합니다.
    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

int main(int argc, char* argv[]) {
    int no_threads;
    int option;
    unsigned char* bitmap;
    Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (argc != 3) {
        printf("> a.out [option] [filename.ppm]\n");
        printf("[option] 0: CUDA, 1~16: OpenMP using 1~16 threads\n");
        printf("for example, '> a.out 8 result.ppm' means executing OpenMP with 8 threads\n");
        exit(0);
    }
    FILE* fp = fopen(argv[2], "w");

    if (strcmp(argv[1], "0") == 0) {
        option = CUDA;
    } else {
        option = OPENMP;
        no_threads = atoi(argv[1]);
    }

    bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    unsigned char* d_bitmap;
    Sphere* d_temp_s;
    hipMalloc((void**)&d_bitmap, sizeof(unsigned char) * DIM * DIM * 4);
    hipMalloc((void**)&d_temp_s, sizeof(Sphere) * SPHERES);

    hipMemcpy(d_temp_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);
    dim3 block(16, 16);
    dim3 grid(DIM / block.x, DIM / block.y);

    hipEventRecord(start, 0);
    kernel<<<grid, block>>>(d_temp_s, d_bitmap);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(bitmap, d_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);

    printf("CUDA ray tracing: %f sec\n", elapsedTime / 1000.0);
    printf("[%s] was generated.\n", argv[2]);

    // 이미지 파일 생성
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", DIM, DIM);
    fprintf(fp, "255\n");
    for (int i = 0; i < DIM * DIM; i++) {
        fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
    }

    fclose(fp);
    hipFree(d_bitmap);
    hipFree(d_temp_s);
    free(bitmap);
    free(temp_s);

    return 0;
}

